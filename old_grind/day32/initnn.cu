#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>

// hyperparameters
#define INPUT_SIZE 784
#define HIDDEN_SIZE 4096
#define OUTPUT_SIZE 10
#define TRAIN_SIZE 10000
#define TEST_SIZE 1000
#define BATCH_SIZE 32
#define EPOCHS 20
#define LEARNING_RATE 0.05

typedef struct {
    float *weights1;
    float *weights2;
    float *bias1;
    float *bias2;
    float *grad_weights1;
    float *grad_weights2;
    float *grad_bias1;
    float *grad_bias2;
} NeuralNetwork;

// cuda check
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(error)); \
            hipDeviceReset(); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)



// cuda kernel for matrix multiplication (A @ B)
__global__ void matmul_a_b_kernel(float *A, float *B, float *C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k) {
        float sum = 0.0f;
        for (int i = 0; i < n; ++i) {
            sum += A[row * n + i] * B[i * k + col];
        }
        C[row * k + col] = sum;
    }
}

// cuda kernel for matrix multiplication (A @ B.T)
__global__ void matmul_a_bt_kernel(float *A, float *B, float *C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k) {
        float sum = 0.0f;
        for (int i = 0; i < n; ++i) {
            sum += A[row * n + i] * B[col * n + i];
        }
        C[row * k + col] = sum;
    }
}

// cuda kernel for matrix multiplication (A.T @ B)
__global__ void matmul_at_b_kernel(float *A, float *B, float *C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < k) {
        float sum = 0.0f;
        for (int i = 0; i < m; ++i) {
            sum += A[i * n + row] * B[i * k + col];
        }
        C[row * k + col] = sum;
    }
}

// cuda kernel for GELU activation
__global__ void gelu_kernel(float *x, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x_val = x[idx];
        float cdf = 0.5f * (1.0f + tanhf(0.7978845608f * (x_val + 0.044715f * x_val * x_val * x_val)));
        x[idx] = x_val * cdf;
    }
}

// cuda kernel for GELU derivative
__global__ void dgelu_kernel(float *x, float *d_gelu_out, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x_val = x[idx];
        float tanh_term = tanhf(0.7978845608f * (x_val + 0.044715f * x_val * x_val * x_val));
        float sech_term = 1.0f / coshf(0.7978845608f * (x_val + 0.044715f * x_val * x_val * x_val));
        float cdf = 0.5f * (1.0f + tanh_term);
        float pdf = 0.5f * sqrtf(2.0f / M_PI) * expf(-0.5f * x_val * x_val);
        d_gelu_out[idx] = cdf + x_val * pdf;
    }
}

// cuda kernel for softmax
__global__ void softmax_kernel(float *x, int batch_size, int size) {
    int b = blockIdx.x;
    if (b < batch_size) {
        float max_val = x[b * size];
        for (int i = 1; i < size; ++i) {
            max_val = fmaxf(max_val, x[b * size + i]);
        }

        float sum = 0.0f;
        for (int i = 0; i < size; ++i) {
            x[b * size + i] = expf(x[b * size + i] - max_val);
            sum += x[b * size + i];
        }

        for (int i = 0; i < size; ++i) {
            x[b * size + i] = fmaxf(x[b * size + i] / sum, 1e-7f);
        }
    }
}