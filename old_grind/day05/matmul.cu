#include<iostream>
#include<hip/hip_runtime.h>


// mat mul kernel
__global__ void matmulkernel(float* A, float* B, float* C, int N) {
    // compute row & col indices for the current thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // ensure thread indices are within matrix bounds
    if (row < N && col < N) {
        float value = 0;
        // dot prod for A row & B col
        for (int i = 0; i < N; ++i) {
            value += A[row * N + i] * B[i * N + col]; 
        }
        // store the computed value in C
        C[row * N + col] = value;
    }
}

// host function
void matmul(float* h_A, float* h_B, float* h_C, int N) {
    int size = N * N * sizeof(float);
    float *d_A, *d_B, *d_C;

    // allocate memory on gpu
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // copy A & B from host to device -> CPU to GPU
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // thread block size 
    dim3 dimBlock(32, 32);
    // grid size to cover all elements of the matrix
    dim3 dimGrid((N + 32 - 1) / 32, (N + 32 - 1) / 32);
    // why? -> even if N is not divisible by 32 it will launch extra blocks as buffer to handle remaining elements

    // for kernel execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    matmulkernel<<<dimGrid, dimBlock>>>(d_C, d_A, d_B, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time = 0;
    hipEventElapsedTime(&time, start, stop);
    std::cout << "Kernel Execution Time: " << time << "ms" << std::endl;

    // copy C back to CPU from GPU
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // free allocated memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    int N = 1024;
    int size = N * N * sizeof(float);

    // allocate pinned memory on the host for better data transfer performance
    float *h_A, *h_B, *h_C;
    hipHostMalloc(&h_A, size, hipHostMallocDefault);
    hipHostMalloc(&h_B, size, hipHostMallocDefault);
    hipHostMalloc(&h_C, size, hipHostMallocDefault);

    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    matmul(h_C, h_A, h_B, N);

    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    return 0;
}
// Kernel Execution Time: 3.74483ms