#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

using data_type = float;

// CUDA error check
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

// cuBLAS error check
#define CUBLAS_CHECK(call) \
    do { \
        hipblasStatus_t status = call; \
        if (status != HIPBLAS_STATUS_SUCCESS) { \
            fprintf(stderr, "CUBLAS error: %d at %s:%d\n", status, __FILE__, __LINE__); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

// utility function to print matrices
void print_matrix(int rows, int cols, const data_type *matrix, int ldm) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%f", matrix[i + j * ldm]);
        }
        printf("\n");
    }
}

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    const int m = 4096;
    const int n = 4096;
    const int k = 4096;
    const int lda = 4096;
    const int ldb = 4096;
    const int ldc = 4096;

    const std::vector<data_type> A(m *k, 1.0f);
    const std::vector<data_type> B(k*n, 1.0f);
    std::vector<data_type> C(m * n);
    const data_type alpha = 1.0f;
    const data_type beta = 0.0f;

    data_type *d_A = nullptr;
    data_type *d_B = nullptr;
    data_type *d_C = nullptr;

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    // create cublas handle, bind a stream
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    // copy data to device
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(data_type) * C.size()));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice,stream));

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start, stream));

    CUBLAS_CHECK(hipblasSgemm(cublasH, transa, transb, m, n, k, &alpha, d_A, lda, d_B, ldb, &beta, d_C, ldc));
    CUDA_CHECK(hipEventRecord(stop, stream));

    CUDA_CHECK(hipEventSynchronize(stop));

    float time = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&time, start, stop));

    CUDA_CHECK(hipMemcpyAsync(C.data(), d_C, sizeof(data_type) * C.size(), hipMemcpyDeviceToHost,stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    double flops = 2.0 * m * n * k;
    double tflops = (flops / (time / 1000.0)) / 1e12;

    printf("CUBLAS GEMM Execution Time: %f ms\n", time);
    printf("CUBLAS GEMM TFLOPs: %f\n", tflops);

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;

}
// CUBLAS GEMM Execution Time: 84.292160 ms
// CUBLAS GEMM TFLOPs: 1.630507