#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

using data_type = double;

// utility function to print a vector
void print_vector(size_t n, const data_type* vec) {
    for (size_t i = 0; i < n; i++) {
        printf("%0.2f ", vec[i]);
    }
    printf("\n");
}

// error checking macro for CUDA API calls
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = (call); \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

// error checking macro for cuBLAS API calls
#define CUBLAS_CHECK(call) \
    do { \
        hipblasStatus_t status = (call); \
        if (status != HIPBLAS_STATUS_SUCCESS) { \
            fprintf(stderr, "cuBLAS error: %d at %s:%d\n", status, __FILE__, __LINE__); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    /*
     *   A = | 1.0 2.0 3.0 4.0 |
     *   B = | 5.0 6.0 7.0 8.0 |
     */

    const std::vector<data_type> A = {1.0, 2.0, 3.0, 4.0};
    const std::vector<data_type> B = {5.0, 6.0, 7.0, 8.0};
    const int incx = 1;
    const int incy = 1;

    data_type result = 0.0;

    data_type *d_A = nullptr;
    data_type *d_B = nullptr;

    printf("A\n");
    print_vector(A.size(), A.data());
    printf("=====\n");

    printf("B\n");
    print_vector(B.size(), B.data());
    printf("=====\n");

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(data_type) * B.size()));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice,
                               stream));

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start, stream));

    /* step 3: compute */
    CUBLAS_CHECK(hipblasDdot(cublasH, A.size(), d_A, incx, d_B, incy, &result));

    CUDA_CHECK(hipEventRecord(stop, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    float time = 0;
    CUDA_CHECK(hipEventElapsedTime(&time, start, stop));

    /*
     *   result = 70.00
     */

    printf("Result\n");
    printf("%0.2f\n", result);
    printf("=====\n");

    printf("Kernel execution time: %f ms\n", time);

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
// Kernel execution time: 0.982880 ms