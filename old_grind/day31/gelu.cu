#include <cmath>
#include <hip/hip_runtime.h>
#include <iostream>

// gelu kernel
__global__ void gelu_kernel(const float* input, float* output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float x = input[idx];
        float cdf = 0.5f * (1.0f + tanhf((0.7978845608028654f * (x + 0.044715f * x * x * x))));
        output[idx] = x * cdf;
    }
}

void gelu(const float* input, float* output, int n) {

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    gelu_kernel<<<numBlocks, blockSize>>>(input, output, n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time = 0.0f;
    hipEventElapsedTime(&time, start, stop);
    std::cout << "Kernel Execution Time: " << time << "ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    int n = 1000000;
    float* h_input = new float[n];
    float* h_output = new float[n];

    for (int i = 0; i < n; ++i) {
        h_input[i] = static_cast<float>(i) - n / 2;
    }

    float *d_input, *d_output;
    hipMalloc((void**)&d_input, n * sizeof(float));
    hipMalloc((void**)&d_output, n * sizeof(float));

    hipMemcpy(d_input, h_input, n * sizeof(float), hipMemcpyHostToDevice);

    gelu(d_input, d_output, n);

    hipMemcpy(h_output, d_output, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    delete[] h_input;
    delete[] h_output;

    return 0;
}
// Kernel Execution Time: 2.90659ms