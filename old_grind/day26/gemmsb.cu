#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

using data_type = float; // fp32 

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    const int m = 4096;
    const int n = 4096;
    const int k = 4096;
    const int lda = 4096;
    const int ldb = 4096;
    const int ldc = 4096;
    const int batch_count = 32;

    const long long int strideA = m * k;
    const long long int strideB = k * n;
    const long long int strideC = m * n;

    // getting values for the matrices
    const std::vector<data_type> A(m * k * batch_count, 1.0f);
    const std::vector<data_type> B(k * n * batch_count, 1.0f);
    std::vector<data_type> C(m * n * batch_count, 0.0f);
    const data_type alpha = 1.0f;
    const data_type beta = 0.0f;

    data_type *d_A = nullptr;
    data_type *d_B = nullptr;
    data_type *d_C = nullptr;

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    hipblasCreate(&cublasH);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

    hipblasSetStream(cublasH, stream);

    hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size());
    hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(data_type) * B.size());
    hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(data_type) * C.size());

    hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_B, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, stream);

    // Sgemm -> single fp precision = fp32
    hipblasSgemmStridedBatched(cublasH, transa, transb, m, n, k, &alpha, d_A, lda, strideA, d_B, ldb, strideB, &beta, d_C, ldc, strideC, batch_count);
    
    hipEventRecord(stop, stream);

    hipEventSynchronize(stop);

    float time = 0;
    hipEventElapsedTime(&time, start, stop);

    hipMemcpyAsync(C.data(), d_C, sizeof(data_type) * C.size(), hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    double flops = 2.0 * m * n * k * batch_count; // FLOPs for the entire batch
    double tflops = (flops / (time / 1000.0)) / 1e12;

    printf("Kernel Execution time: %f ms\n", time);
    printf("TFLOPs: %f\n", tflops);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipblasDestroy(cublasH);

    hipStreamDestroy(stream);

    hipDeviceReset();

    return EXIT_SUCCESS;
}
// for batch count = 64
// Kernel Execution time: 86.794205 ms
// TFLOPs: 101.344244

// for batch count = 32
// Kernel Execution time: 144.666046 ms
// TFLOPs: 30.401374