#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>
#include <torch/types.h>
#include <vector>

#define WARP_SIZE 32

#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func)                                   \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                                   \
  if (((T).options().dtype() != (th_type))) {                                  \
    std::cout << "Tensor Info:" << (T).options() << std::endl;                 \
    throw std::runtime_error("values must be " #th_type);                      \
  }


// Swish x: N, y: N y=x*sigmoid(x)
__device__ __forceinline__ float swish(float x) {
  return x / (1.0f + expf(-x));
}

__device__ __forceinline__ half swish_half(half x) {
  return __hmul(x, __hdiv(__float2half(1.0f),
                          __hadd(__float2half(1.0f), hexp(__hneg(x)))));
}

__global__ void swish_f32_kernel(float *x, float *y, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N)
    y[idx] = swish(x[idx]);
}

__global__ void swish_f16_kernel(half *x, half *y, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N)
    y[idx] = swish_half(x[idx]);
}

#define TORCH_BINDING_SWISH(packed_type, th_type, element_type, n_elements)    \
  void swish_##packed_type(torch::Tensor x, torch::Tensor y) {                 \
    CHECK_TORCH_TENSOR_DTYPE(x, (th_type))                                     \
    CHECK_TORCH_TENSOR_DTYPE(y, (th_type))                                     \
    const int ndim = x.dim();                                                  \
    if (ndim != 2) {                                                           \
      int N = 1;                                                               \
      for (int i = 0; i < ndim; ++i) {                                         \
        N *= x.size(i);                                                        \
      }                                                                        \
      dim3 block(256 / (n_elements));                                          \
      dim3 grid((N + 256 - 1) / 256);                                          \
      swish_##packed_type##_kernel<<<grid, block>>>(                           \
          reinterpret_cast<element_type *>(x.data_ptr()),                      \
          reinterpret_cast<element_type *>(y.data_ptr()), N);                  \
    } else {                                                                   \
      const int S = x.size(0);                                                 \
      const int K = x.size(1);                                                 \
      const int N = S * K;                                                     \
      if ((K / (n_elements)) <= 1024) {                                        \
        dim3 block(K / (n_elements));                                          \
        dim3 grid(S);                                                          \
        swish_##packed_type##_kernel<<<grid, block>>>(                         \
            reinterpret_cast<element_type *>(x.data_ptr()),                    \
            reinterpret_cast<element_type *>(y.data_ptr()), N);                \
      } else {                                                                 \
        int N = 1;                                                             \
        for (int i = 0; i < ndim; ++i) {                                       \
          N *= x.size(i);                                                      \
        }                                                                      \
        dim3 block(256 / (n_elements));                                        \
        dim3 grid((N + 256 - 1) / 256);                                        \
        swish_##packed_type##_kernel<<<grid, block>>>(                         \
            reinterpret_cast<element_type *>(x.data_ptr()),                    \
            reinterpret_cast<element_type *>(y.data_ptr()), N);                \
      }                                                                        \
    }                                                                          \
  }

TORCH_BINDING_SWISH(f32, torch::kFloat32, float, 1)
TORCH_BINDING_SWISH(f16, torch::kHalf, half, 1)

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  TORCH_BINDING_COMMON_EXTENSION(swish_f32)
  TORCH_BINDING_COMMON_EXTENSION(swish_f16)
}